#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"

__global__ void add(int a, int b, int *c) { *c = a + b; }

int main() {
  int c;
  int *d_c;

  hipMalloc((void **)&d_c, sizeof(int));

  add << <1, 1>>> (3, 5, d_c);

  hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

  printf("result is  %d\n", c);

  hipFree(d_c);
}
