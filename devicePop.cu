
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"


int main()
{




hipDeviceProp_t prop;

int count;

hipGetDeviceCount(&count);

for(int i=0;i<count;i++)
{

hipGetDeviceProperties(&prop,i);

printf("=======================================");
printf("name= %d\n",prop.name);
printf(" compute cap= %d\n",prop.major);
printf(" GlobalMem = %d\n",prop.totalGlobalMem);
printf(" multi peocess count = %d\n",prop.multiProcessorCount);
printf(" max thread/block = %d\n",prop.maxThreadsPerBlock);



}


}

                       
