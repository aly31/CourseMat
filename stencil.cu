#include "hip/hip_runtime.h"
/*
Program: addVector

A basic C/CUDA code to get started with GPU computing
The program check for GPUs on the host, prints some of the specifications
of the GPU, and then set a GPU with certain specs for computation. The CUDA
kernel illustrates allocating memory on the device, copying data to the device
and then doing a simple addition on the GPU and copying the results back to the
host and finally freeing the memory on the device.

Author: Inanc Senocak

to compile: nvcc -O2 addVectorCUDA.cu -o run.exe
to execute: ./run.exe

Demonstrate stack vs. heap memory by making NX small and large values

*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <sys/time.h>
#define BLOCKSIZE 256
#define RADIUS 3
#define NX 10000000 // make NX a large number to test stack vs. heap
#define LINSIZE BLOCKSIZE + 2 * RADIUS

/* The argument now should be a double (not a pointer to a double) */
#define GET_TIME(now)                                                          \
  {                                                                            \
    struct timeval t;                                                          \
    gettimeofday(&t, NULL);                                                    \
    now = t.tv_sec + t.tv_usec / 1000000.0;                                    \
  }

typedef float REAL;
__global__ void stencil_1d(int *in, int *out) {
  __shared__ int temp[LINSIZE];

  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + RADIUS;

  // Read input elements into shared memory

  temp[lindex] = in[gindex];
  __syncthreads();

  // Apply the stencil
  int result = 0;
  for (int offset = -RADIUS; offset <= RADIUS; offset++)
    result += temp[lindex + offset];

  // Store the result
  out[gindex] = result;
}

// stencil for CPU
void cpu_stencil(int *in, int *out) {
  for (int i = RADIUS; i < NX; i++) {
    REAL sum = 0.0f;
    for (int j = -RADIUS; j <= RADIUS; j++) {
      sum += in[i + j];
    }
    out[i] = sum;
  }
}
int main(void) {
  /*
    // let's see how many CUDA capable GPUs we have

    int gpuCount;

    hipGetDeviceCount(&gpuCount);

    printf(" Number of GPUs = %d\n", gpuCount);

    int myDevice = 0;

    hipSetDevice(myDevice);
  */

  // let's use the device to do some calculations
  //  int a[NX],b[NX],c[NX];
  /*
    int *a = malloc(NX * sizeof(*a));
    int *b = malloc(NX * sizeof(*b));
    int *c = malloc(NX * sizeof(*c));
  */
  int *a = (int *)malloc(NX * sizeof(*a));
  int *b = (int *)malloc(NX * sizeof(*b));
  int *c = (int *)malloc(NX * sizeof(*c));

  int *d_a, *d_b, *d_c; // create pointers for the device

  hipMallocManaged(&d_a, NX * sizeof(int)); // attach d_ to varibles
  hipMallocManaged(&d_b, NX * sizeof(int));
  hipMallocManaged(&d_c, NX * sizeof(int));

  // Let's fill the arrays with some numbers

  for (int i = 0; i < NX; i++) {
    a[i] = 4;
    b[i] = 0;
    c[i] = 0;
  }

  // Let's create the infrastructure to time the host & device operations

  double start, finish; // for the CPU

  hipEvent_t timeStart,
      timeStop; // WARNING!!! use events only to time the device
  hipEventCreate(&timeStart);
  hipEventCreate(&timeStop);
  float elapsedTime; // make sure it is of type float, precision is
                     // milliseconds (ms) !!!

  GET_TIME(start);
  // Let's do the following operation on the arrays on the host: a = b +
  // c
  cpu_stencil(a, b);

  GET_TIME(finish);

  printf("elapsed wall time (host) = %.6f seconds\n", finish - start);

  // Let's print the results on the screen

  printf("b, c, a=b+c\n");

  //     for (int i=0; i<NX; i++) {
  //         printf("%d %2d %3d\n", b[i], c[i], a[i]);
  //     }

  hipMemcpy(d_a, a, NX * sizeof(float),
             hipMemcpyHostToDevice); // memcpy(dest,src,...

  hipEventRecord(timeStart, 0); // don't worry for the 2nd argument zero, it is
                                 /*                    // about cuda streams

    dim3 threadsPerBlock(16,
                         16); // Best practice of having 256 threads per block
    dim3 numBlocks((NX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (NY + threadsPerBlock.y - 1) / threadsPerBlock.y);
  */
  int nBlocks = (NX + BLOCKSIZE - 1) / BLOCKSIZE; // round up if n
  stencil_1d << <nBlocks, BLOCKSIZE>>>
      (d_a, d_b); // Be careful with the syntax!

  //*/
  /*     int blockSize = 256;
       int nBlocks   = (NX + blockSize -1) / blockSize; //round up if n
     is not a multiple of blocksize myKernel <<<nBlocks,
     blockSize>>>(d_a, d_b, d_c);
       //myKernel<<<1, 1>>> (d_a, d_b, d_c);*/

  printf("a[100] = %4d\n", a[100]);

  hipEventRecord(timeStop, 0);
  hipEventSynchronize(timeStop);

  // WARNING!!! do not simply print (timeStop-timeStart)!!

  hipEventElapsedTime(&elapsedTime, timeStart, timeStop);

  printf("elapsed wall time (device) = %3.1f ms\n", elapsedTime);

  hipEventDestroy(timeStart);
  hipEventDestroy(timeStop);

  hipMemcpy(c, d_b, NX * sizeof(float), hipMemcpyDeviceToHost);

  //     for (int i=0; i<NX; i++) {
  //         printf("%3d\n", a[i]);
  //     }

  hipFree(d_a);
  hipFree(d_b);
  free(a);
  free(b);
  free(c);
  return EXIT_SUCCESS;
}
