
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"


int main()
{




hipDeviceProp_t prop;

int count;

hipGetDeviceCount(&count);

for(int i=0;i<count;i++)
{

hipGetDeviceProperties(&prop,i);

printf("=======================================\n");
printf(" name= %s\n",prop.name);
printf(" compute cap= %d\n",prop.major);
printf(" GlobalMem = %d\n",prop.totalGlobalMem/1000000000);
printf(" multi process count = %d\n",prop.multiProcessorCount);
printf(" max thread/block = %d\n",prop.maxThreadsPerBlock);

}


}

                       
