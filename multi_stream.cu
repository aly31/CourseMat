
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"

//  nvprof --print-gpu-trace ./myEx (0 or 1) 
//  see the kernel launch time
//  Modified from NVIDIA example to illustrate streams 
// enter 0 and run
// enter 1 and run again
// compare the start times for the two kernels

const int N=1 << 26;

__global__ void kernel1(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

__global__ void kernel2(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(1.2,i));
    }
}

int main(int argc, char *argv[])
{
    const int num_streams = 2;
    printf(" N =% d\n",N);

    hipStream_t streams[num_streams];
    float *data[num_streams];


   if(argc!=2)
   {
   printf("enter 0 for default tream and 1 for multi stream as command line arg\n");
   exit(1);
}

   int bol=atoi(argv[1]);

    for (int i = 0; i < num_streams; i++) {

        hipStreamCreate(&streams[i]);

        hipMalloc(&data[i], N * sizeof(float));

    }


if(bol==0)
{
        hipMalloc(&data[0], N * sizeof(float));
        kernel1<<<1, 64, 0>>>(data[0], N);
        hipMalloc(&data[1], N * sizeof(float));
        kernel2<<<1, 64, 0>>>(data[1], N);

 }
else
{
        kernel1<<<1, 64, 0, streams[0]>>>(data[0], N);
         kernel2<<<1, 64, 0, streams[1]>>>(data[1], N);

}




    hipDeviceReset();

    return 0;
}


