#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

// __device__ functions may only be called from __global__ functions or other
// __device__ functions.  Unlike __global__ functions, __device__ functions are
// not configured, and have no restriction on return type.

__device__ int get_constant(void) {
  // just return 7
  return 9;
}

__device__ int get_block_index(void) {
  // return the index of the current thread's block
  return blockIdx.x;
}

__device__ int get_thread_index(void) {

  // return the index of the current thread within its block

  return threadIdx.x;
}

__device__ int get_global_index(void) {
  // return the index of the current thread across the entire grid launch
  return blockIdx.x * blockDim.x + threadIdx.x;
}

// kernel1 returns the result of calling the __device__ function
// return_constant():

__global__ void kernel1(int *array) {
  int index = get_global_index();
  array[index] = get_constant();
}

// kernel2 returns the result of calling the __device__ function
// return_block_index():

__global__ void kernel2(int *array) {
  int index = get_global_index();
  array[index] = get_block_index();
}
__global__ void kernel3(int *array) {
  int index = get_global_index();
  array[index] = get_thread_index();
}

int main(void) {
  int num_elements = 256;
  int num_bytes = num_elements * sizeof(int);
  int *device_array = NULL;
  int *host_array = NULL;
  // malloc a host array
  host_array = (int *)malloc(num_bytes);

  // hipMalloc a device array

  hipMalloc((void **)&device_array, num_bytes);

  // if either memory allocation failed, report an error message

  if (host_array == NULL || device_array == NULL) {

    printf("couldn't allocate memory\n");

    return 1;
  }
  // choose a launch configuration

  int block_size = 128;

  int grid_size = num_elements / block_size;

  // launch each kernel and print out the results

  kernel1 << <grid_size, block_size>>> (device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel1 results:\n");

  for (int i = 0; i < num_elements; ++i) {

    printf("%d ", host_array[i]);
  }

  printf("\n\n");

  kernel2 << <grid_size, block_size>>> (device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel2 results:\n");

  for (int i = 0; i < num_elements; ++i) {
    printf("%d ", host_array[i]);
  }

  printf("\n\n");

  kernel3 << <grid_size, block_size>>> (device_array);

  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel3 results:\n");

  for (int i = 0; i < num_elements; ++i) {

    printf("%d ", host_array[i]);
  }

  // deallocate memory

  free(host_array);

  hipFree(device_array);
}
