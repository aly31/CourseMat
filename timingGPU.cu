#include "stdio.h"
#include "stdlib.h"
#include <hip/hip_runtime.h>



// find two minor bugs in this code

__global__ void mykernel(void) {
printf("Hello World from GPU!\n");
}
int main(void) {

float time;

hipEvent_t t1, t2;

hipEventCreate(&t1);
hipEventCreate(&t2);

// start the timing
hipEventRecord(t1);

mykernel<<<1,1>>>();

//end timing
hipEventRecord(t2);

// use this function, do not etract t1 from t2
// time is in milliseconds

hipEventElapsedTime(&time, t1, t2);

//synch 
hipEventSynchronize(t2);


hipDeviceSynchronize();


// what is missig? 

printf("time=%lf milli seconds\n",time);

return 0;
}

