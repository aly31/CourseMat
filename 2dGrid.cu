#include "hip/hip_runtime.h"
// This example demonstrates how to launch two dimensional grids of CUDA
// threads.

#include <stdlib.h>
#include <stdio.h>

__global__ void kernel(int *array) {
  // compute the two dimensional index of this particular
  // thread in the grid
  // do the usual computation separately in each dimension:

  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  // use the two 2D indices to compute a single linear index
  // fill in grid Dim as I explained in the class

  int grid_width = xxx;

  // index mapps y to a global 1 D
  int index = ;

  // use the two 2D block indices to compute a single linear block index

  int result = blockIdx.y * gridDim.x + blockIdx.x;

  // write out the result
  array[index] = result;
}

int main(void) {

  int num_elements_x = 16;

  int num_elements_y = 16;

  int num_bytes = num_elements_x * num_elements_y * sizeof(int);

  int *device_array = 0;

  int *host_array = 0;

  // malloc a host array

  host_array = (int *)malloc(num_bytes);

  // hipMalloc a device array

  hipMalloc((void **)&device_array, num_bytes);

  // if either memory allocation failed, report an error message

  if (host_array == 0 || device_array == 0) {

    printf("couldn't allocate memory\n");

    return 1;
  }

// choose a two dimensional launch configuration

// use the dim3 type when launches are not one dimensional
