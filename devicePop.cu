
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"


int main()
{




hipDeviceProp_t prop;

int count;

hipGetDeviceCount(&count);

for(int i=0;i<count;i++)
{

hipGetDeviceProperties(&prop,i);

printf("=======================================\n");
printf(" name= %s\n",prop.name);
printf(" compute cap= %d\n",prop.major);
printf(" GlobalMem = %d\n",prop.totalGlobalMem);
printf(" multi process count = %ld\n",prop.multiProcessorCount);
printf(" max thread/block = %d\n",prop.maxThreadsPerBlock);



}


}

                       
