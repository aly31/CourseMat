
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"


__global__ void vecAdd(double *a, double *b, double *c, int n)

{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n) 
        c[id] = a[id] + b[id];
}


int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;

    // Host input vectors
    double *a=NULL;
    double *b=NULL;
    //Host output vector
    double *c=NULL;

    // Device input vectors
    double *d_a=NULL;
    double *d_b=NULL;
    //Device output vector
    double *d_c=NULL;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);

    // Allocate memory for each vector on host
    a = (double*)malloc(n*sizeof(double));
    b = (double*)malloc(n*sizeof(double));
    c = (double*)malloc(n*sizeof(double));

    // Allocate memory for each vector on GPU, including d_a, d_b and d_c
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);


    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        a[i] = sin(i)*sin(i);
        b[i] = cos(i)*cos(i);
    }

    // Copy host vectors to device
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx
   hipMemcpy( d_a, a, n*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy( d_b, b, n*sizeof(double), hipMemcpyHostToDevice);
 

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx
    // xxxxxxxxxxxxxxxxxxxxxxxxxxxxx
     hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
 for(i=0; i<n; i++)
        sum += c[i];
    printf("final result: %f\n", sum/n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(a);
    free(b);
    free(c);

    return 0;
}

