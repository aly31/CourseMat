
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"


__global__ void add(int a, int b, int *c)
{
*c=a+b;
}


int main()
{
//int c;
int *d_c;

hipMallocManaged((void**)&d_c, sizeof(int));

add<<<1,1>>>(3,5,d_c);

hipDeviceSynchronize();

printf("result is  %d\n", *d_c);

hipFree(d_c);


}


